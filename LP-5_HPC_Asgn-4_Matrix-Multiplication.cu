
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 3
__global__ void matrixMultiplication(float *A, float *B, float *C, int n)
{
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
if (i < n && j < n) {
float sum = 0.0f;
for (int k = 0; k < n; ++k) {
sum += A[i * n + k] * B[k * n + j];
}
C[i * n + j] = sum;
}
}
int main()
{
float A[N][N] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
float B[N][N] = {{9, 8, 7}, {6, 5, 4}, {3, 2, 1}};
float C[N][N] = {0};
// Allocate device memory
float *d_A, *d_B, *d_C;
hipMalloc(&d_A, N * N * sizeof(float));
hipMalloc(&d_B, N * N * sizeof(float));
hipMalloc(&d_C, N * N * sizeof(float));
// Copy input matrices from host to device
hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);
// Set the grid and block dimensions
dim3 gridDim(ceil(N/16.0), ceil(N/16.0), 1);
dim3 blockDim(16, 16, 1);
// Launch the kernel
matrixMultiplication<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
// Copy result matrix from device to host
hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
// Print the result matrix
printf("Result Matrix:\n");
for (int i = 0; i < N; ++i) {
for (int j = 0; j < N; ++j) {
printf("%.1f ", C[i][j]);
}
printf("\n");
}
// Free device memory
hipFree(d_A);
hipFree(d_B);
hipFree(d_C);
return 0;
}